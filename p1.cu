
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define n 1024

__global__ void mul_matrix(int *a, int *b, int *c) {
    int my_x, my_y;
    my_x = blockIdx.x * blockDim.x + threadIdx.x;
    my_y = blockIdx.y * blockDim.y + threadIdx.y;

    int local_c = 0;
    for (int i = 0; i < n; i++) {
        local_c += a[my_x * n + i] * b[i * n + my_y];
    }
    c[my_x * n + my_y] = local_c; 
}

int main() {
    int i;
    int *a = (int*)malloc(sizeof(int) * n * n);
    int *b = (int*)malloc(sizeof(int) * n * n);
    int *c = (int*)malloc(sizeof(int) * n * n); 

    dim3 dimGrid(64, 64);
    dim3 dimBlock(16, 16);

    for (i = 0; i < n * n; i++) {
        a[i] = 1;
        b[i] = 2;
    }

    int *gpu_a, *gpu_b, *gpu_c;
    hipMalloc((void**)&gpu_a, n * n * sizeof(int));
    hipMalloc((void**)&gpu_b, n * n * sizeof(int));
    hipMalloc((void**)&gpu_c, n * n * sizeof(int));

    hipMemcpy(gpu_a, a, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, n * n * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    mul_matrix<<<dimGrid, dimBlock>>>(gpu_a, gpu_b, gpu_c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Kernel Execution Time: %f ms\n", milliseconds);

    hipMemcpy(c, gpu_c, sizeof(int) * n * n, hipMemcpyDeviceToHost);

    printf("Result at C[451][451]: %d\n", c[451 * n + 451]);

    free(a);
    free(b);
    free(c); 
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    return 0;
}
